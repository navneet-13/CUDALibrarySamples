#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hipblaslt.h>
#include <iostream>

#include "sample_cublasLt_LtHSHgemmStridedBatchSimple.h"
#include "helpers.h"

/// Sample wrapper executing mixed precision gemm with hipblasLtMatmul, nearly a drop-in replacement for hipblasGemmEx,
/// with addition of the workspace to support split-K algorithms
///
/// pointer mode is always host, to change it configure the appropriate matmul descriptor attribute
/// matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed
void LtHSHgemmStridedBatchSimple(hipblasLtHandle_t ltHandle,
                                 hipStream_t stream,
                                 hipblasOperation_t transa,
                                 hipblasOperation_t transb,
                                 size_t m,
                                 size_t n,
                                 size_t k,
                                 const float *alpha, /* host pointer */
                                 const __half *A,
                                 int lda,
                                 int64_t stridea,
                                 const __half *B,
                                 int ldb,
                                 int64_t strideb,
                                 const float *beta, /* host pointer */
                                 __half *C,
                                 int ldc,
                                 int64_t stridec,
                                 int batchCount,
                                 void *workspace,
                                 size_t workspaceSize) {

    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create matrix descriptors, we need to configure batch size and counts in this case
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridea, sizeof(stridea)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &strideb, sizeof(strideb)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16F, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT, &batchCount, sizeof(batchCount)));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET, &stridec, sizeof(stridec)));

    // in this simplified example we take advantage of hipblasLtMatmul shortcut notation with algo=NULL which will force
    // matmul to get the basic heuristic result internally. Downsides of this approach are that there is no way to
    // configure search preferences (e.g. disallow tensor operations or some reduction schemes) and no way to store the
    // algo for later use

    hipblasLtMatmulPreference_t preference = NULL;
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));

    // Set max workspace size allowed
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(
        preference,
    HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
    &workspaceSize, sizeof(workspaceSize)));

    const int requestAlgoCount = 10;
    hipblasLtMatmulHeuristicResult_t heuristicResults[requestAlgoCount];
    int returnedAlgoCount = 0;

    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(
        ltHandle,
        operationDesc,
        Adesc,
        Bdesc,
        Cdesc,
        Cdesc,
        preference,
        requestAlgoCount,
        heuristicResults,
        &returnedAlgoCount));

    if (returnedAlgoCount == 0) {
        std::cerr << "No suitable algorithm found.\n";
        return;
    }

    // Choose best one (usually first one)
    auto bestAlgo = &heuristicResults[0].algo;

    

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);

    // // hipStream_t stream;
    // hipStreamCreate(&stream);
    // // hipblasSetStream(ltHandle, stream);

    // hipEventRecord(start, stream);

    // hipEventRecord(start,0);

    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     operationDesc,
                                     alpha,
                                     A,
                                     Adesc,
                                     B,
                                     Bdesc,
                                     beta,
                                     C,
                                     Cdesc,
                                      C,
                                     Cdesc,
                                     bestAlgo,
                                     workspace,
                                     workspaceSize,
                                     stream));

// hipEventRecord(stop,0);
    // hipEventRecord(stop, stream);
    // hipEventSynchronize(stop);
    // hipStreamDestroy(stream);
    // hipEventRecord(stop,0);
    // hipEventSynchronize(stop);

    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    // hipEventDestroy(start);
    // hipEventDestroy(stop);

    // descriptors are no longer needed as all GPU work was already enqueued
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));
}
